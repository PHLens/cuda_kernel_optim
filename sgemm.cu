#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "kernels.cuh"
#include "utils.cuh"

#define MY_GEMM sgemm1_1

int main(int argc, const char* argv[]) {
  if (argc != 4) {
    printf("Arguments error! Usage: ./sgemm_gpu [M] [N] [K]\n");
    exit(0);
  }

  size_t M = atoi(argv[1]);
  size_t N = atoi(argv[2]);
  size_t K = atoi(argv[3]);

  size_t size_A = M * K * sizeof(float);
  size_t size_B = K * N * sizeof(float);
  size_t size_C = M * N * sizeof(float);
  float* A = (float*)malloc(size_A);
  float* B = (float*)malloc(size_B);
  float* C = (float*)malloc(size_C);
  float* C1 = (float*)malloc(size_C);

  const int BLOCK_SIZE_M = 32;
  const int BLOCK_SIZE_N = 32;
  const int BLOCK_SIZE_K = 32;

  float *d_A, *d_B, *d_C, *d_C1;
  checkCudaErrors(hipMalloc(&d_A, size_A));
  checkCudaErrors(hipMalloc(&d_B, size_B));
  checkCudaErrors(hipMalloc(&d_C, size_C));
  checkCudaErrors(hipMalloc(&d_C1, size_C));

  // generate data
  for (int i = 0; i < M * K; i++) {
    A[i] = i / 13;
  }
  for (int i = 0; i < K * N; i++) {
    B[i] = i % 13;
  }

  checkCudaErrors(hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C, C, size_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_C1, C1, size_C, hipMemcpyHostToDevice));

  hipEvent_t s, e;
  checkCudaErrors(hipEventCreate(&s));
  checkCudaErrors(hipEventCreate(&e));
  float total_time_ms = 0.0;
  int nIters = 100;

  checkCudaErrors(hipEventRecord(s));

  for (int i = 0; i < nIters; i++) {
    // Define the block size and grid size
    dim3 blockDim(BLOCK_SIZE_N, BLOCK_SIZE_M);
    dim3 gridDim((N + BLOCK_SIZE_N - 1) / BLOCK_SIZE_N, (M + BLOCK_SIZE_M - 1) / BLOCK_SIZE_M);

    MY_GEMM<BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K><<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
  }

  checkCudaErrors(hipEventRecord(e));
  checkCudaErrors(hipEventSynchronize(e));
  checkCudaErrors(hipEventElapsedTime(&total_time_ms, s, e));
  checkCudaErrors(hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost));

  printf("My kernel average time: %f ms.\n", total_time_ms / nIters);

  // cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0;
  float beta = 0.0;

  checkCudaErrors(hipEventRecord(s));

  for (int i = 0; i < nIters; i++) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C1, N);
  }

  checkCudaErrors(hipEventRecord(e));
  checkCudaErrors(hipEventSynchronize(e));
  checkCudaErrors(hipEventElapsedTime(&total_time_ms, s, e));
  checkCudaErrors(hipMemcpy(C1, d_C1, size_C, hipMemcpyDeviceToHost));

  printf("CuBlas kernel average time: %f ms.\n", total_time_ms / nIters);

  hipblasDestroy(handle);

  double eps = 1.e-6;
  isEqualT(C, C1, M, N, eps);

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipFree(d_C1));
  checkCudaErrors(hipEventDestroy(s));
  checkCudaErrors(hipEventDestroy(e));

  free(A);
  free(B);
  free(C);
  free(C1);
  return 0;
}
