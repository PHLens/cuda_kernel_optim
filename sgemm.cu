#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "kernels.cuh"
#include "utils.cuh"

#define MY_GEMM sgemm0

int main(int argc, const char* argv[]) {
  if (argc != 4) {
    printf("Arguments error! Usage: ./sgemm_gpu [M] [N] [K]\n");
    exit(0);
  }

  size_t M = atoi(argv[1]);
  size_t N = atoi(argv[2]);
  size_t K = atoi(argv[3]);

  size_t size_A = M * K * sizeof(float);
  size_t size_B = K * N * sizeof(float);
  size_t size_C = M * N * sizeof(float);
  float* A = (float*)malloc(size_A);
  float* B = (float*)malloc(size_B);
  float* C = (float*)malloc(size_C);
  float* C1 = (float*)malloc(size_C);

  float *d_A, *d_B, *d_C, *d_C1;
  hipMalloc(&d_A, size_A);
  hipMalloc(&d_B, size_B);
  hipMalloc(&d_C, size_C);
  hipMalloc(&d_C1, size_C);

  // generate data
  for (int i = 0; i < M * K; i++) {
    A[i] = i + 1;
  }
  for (int i = 0; i < K * N; i++) {
    B[i] = i + 1;
  }
  // printf("A: ");
  // printM(A, M, K);
  // printf("B: ");
  // printM(B, K, N);

  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, size_C, hipMemcpyHostToDevice);
  hipMemcpy(d_C1, C1, size_C, hipMemcpyHostToDevice);

  hipEvent_t s, e;
  hipEventCreate(&s);
  hipEventCreate(&e);
  float total_time_ms = 0.0;
  int nIters = 10;

  hipEventRecord(s);

  for (int i = 0; i < nIters; i++) {
    // Define the block size and grid size
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    MY_GEMM<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
  }

  hipEventRecord(e);
  hipEventSynchronize(e);
  hipEventElapsedTime(&total_time_ms, s, e);
  hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

  // printf("C: ");
  // printM(C, M, N);
  printf("My kernel average time: %f ms.\n", total_time_ms / nIters);

  // cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0;
  float beta = 0.0;
  hipEventRecord(s);
  for (int i = 0; i < nIters; i++) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C1, N);
  }

  hipEventRecord(e);
  hipEventSynchronize(e);
  hipEventElapsedTime(&total_time_ms, s, e);
  hipMemcpy(C1, d_C1, size_C, hipMemcpyDeviceToHost);

  // printf("C1: ");
  // printM(C1, M, N);
  printf("CuBlas kernel average time: %f ms.\n", total_time_ms / nIters);

  hipblasDestroy(handle); 

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_C1);
  hipEventDestroy(s);
  hipEventDestroy(e);

  free(A);
  free(B);
  free(C);
  free(C1);
  return 0;
}
